#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define BLOCK_SIZE 512

#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) {
      fprintf(stdout,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define SET_SIZE 64
#define LINE_SIZE 128
#define STRIDE LINE_SIZE*SET_SIZE

__global__ void kernel(char *out, char *in, int size) {
    long tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) out[tid] = in[(tid*STRIDE)%size];
}

void benchmark(char *out_host, char *in_host, int size) {
    char *out_gpu, *in_gpu;

    // Allocate arrays in GPU memory
    checkCuda(hipMalloc((void**)&(out_gpu), size*sizeof(char)));
    checkCuda(hipMalloc((void**)&(in_gpu), size*sizeof(char)));

    // Copy input to GPU
    checkCuda(hipMemcpy(in_gpu, in_host, size*sizeof(char), hipMemcpyHostToDevice));

    dim3 dimGrid(1+(size-1)/BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE);

    // Execute kernel
    kernel<<<dimGrid, dimBlock>>>(out_gpu, in_gpu, size);

    // Print any errors that may have occured in kernel
    checkCuda(hipPeekAtLastError());

    // Retrieve results from the GPU
    checkCuda(hipMemcpy(out_host, out_gpu, size*sizeof(char), hipMemcpyDeviceToHost));

    // Free resources and end the program
    checkCuda(hipFree(out_gpu));
    checkCuda(hipFree(in_gpu));
}
