
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime_api.h>
// limited version of checkCudaErrors from helper_cuda.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x*3 + i*3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}

int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = 3*num_pixels*sizeof(float);


    float *fb_gpu;
    // checkCudaErrors(cudaMallocManaged((void **)&fb_gpu, fb_size));  // not supported in CUDA 4
    checkCudaErrors(hipMalloc((void **)&fb_gpu, fb_size));
    float *fb_host = (float*) malloc(fb_size);

    clock_t start, stop;
    start = clock();

    dim3 blocks(nx/tx+1,ny/ty+1);

    dim3 threads(tx,ty);
    // cudaProfilerStart();  // not supported in CUDA 4
    render<<<blocks, threads>>>(fb_gpu, nx, ny);
    // cudaProfilerStop();  // not supported in CUDA 4
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    checkCudaErrors(hipMemcpy(fb_host, fb_gpu, fb_size, hipMemcpyDeviceToHost));
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*3*nx + i*3;
            float r = fb_host[pixel_index + 0];
            float g = fb_host[pixel_index + 1];
            float b = fb_host[pixel_index + 2];
            int ir = int(255.99*r);
            int ig = int(255.99*g);
            int ib = int(255.99*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb_gpu));
    free(fb_host);
}
